#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "mm_kernel.h"

__global__ void mat_mul_kernel(int m, int n, int p, float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int k;

  // Only let this thread compute if it is in C
  if(row < p && col < m) {
    C[row*p+col] = 0;
    for(k=0; k<n; k++){
      C[row*p+col] += A[row*n+k]*B[k*p+col];
    } 
  }
}

void matrix_mult(int m, int n, int p, float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  float* GPU_A, *GPU_B, *GPU_C;
  int size_a = (m*n),
      size_b = (n*p),
      size_c = (m*p);

  // Alloc and copy GPU memory
  hipMalloc((void **)&GPU_A, sizeof(float) * size_a);
  hipMalloc((void **)&GPU_B, sizeof(float) * size_b);
  hipMalloc((void **)&GPU_C, sizeof(float) * size_c);

  hipMemcpy(GPU_A, A, sizeof(float) * size_a, hipMemcpyHostToDevice);
  hipMemcpy(GPU_B, B, sizeof(float) * size_b, hipMemcpyHostToDevice);
  
  dim3 threadsPerBlock(32,16);
  dim3 blocksPerGrid(1, 1);
  if (size_c > 512){
    blocksPerGrid.x = ceil(double(m)/32);
    blocksPerGrid.y = ceil(double(p)/16);
  }

  mat_mul_kernel<<<blocksPerGrid,threadsPerBlock>>>(m,n,p,GPU_A,GPU_B,GPU_C);

  hipDeviceSynchronize();

  // Copy the data back to the host
  hipMemcpy(C, GPU_C, sizeof(float) * size_c, hipMemcpyDeviceToHost);

  hipFree(GPU_A);
  hipFree(GPU_B);
  hipFree(GPU_C);
} 
