#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mm_kernel.h"

__global__ void mat_mul(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  C[index] = A[index]*B[index];

  // for(i=0; i<m; i++) {
  //   for(l=0; l<p; l++){
  //     C[i*p+l]=0;
  //   }
  //   for(k=0; k<n; k++) {
  //     for(j=0; j<p; j++) {
  //       C[i*p+j] += A[i*n+k]*B[k*p+j];
  //     }
  //   }
  // }
}

void matrix_mult(int m, int n, int p, float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int i, j, k, l;
  int size_a = (m*n),
      size_b = (n*p),
      size_c = (m*p);

  // Alloc and copy GPU memory
  hipMalloc((void **)&GPU_A, sizeof(float) * size_a);
  hipMalloc((void **)&GPU_B, sizeof(float) * size_b);
  hipMalloc((void **)&GPU_C, sizeof(float) * size_c);

  hipMemcpy(GPU_A, A, sizeof(float) * size_a, hipMemcpyHostToDevice);
  hipMemcpy(GPU_B, B, sizeof(float) * size_b, hipMemcpyHostToDevice);

  mat_mul<<<size_c, 512>>>(GPU_A, GPU_B, GPU_C);

  hipDeviceSynchronize();

  // Copy the data back to the host
  hipMemcpy(C, GPU_C, sizeof(float) * size_c, hipMemcpyDeviceToHost);

  hipFree(GPU_A);
  hipFree(GPU_B);
  hipFree(GPU_C);
} 
