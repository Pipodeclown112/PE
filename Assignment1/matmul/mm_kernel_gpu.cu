#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "mm_kernel.h"

__global__ void mat_mul_kernel(int m, int n, int p, float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int k;

  printf("HELLO");

  // Only let this thread compute if it is in C
  if(row < m && col < p) {
    C[row*p+col] = 0;
    for(k=0; k<n; k++){
      C[row*p+col] += A[row*n+k]*B[k*p+col];
    } 
  }
}

void matrix_mult(int m, int n, int p, float* A, float* B, float* C) {
  float* GPU_A, *GPU_B, *GPU_C;
  int size_a = (m*n),
      size_b = (n*p),
      size_c = (m*p);

  // Alloc and copy GPU memory
  hipMalloc((void **)&GPU_A, sizeof(float) * size_a);
  hipMalloc((void **)&GPU_B, sizeof(float) * size_b);
  hipMalloc((void **)&GPU_C, sizeof(float) * size_c);

  hipMemcpy(GPU_A, A, sizeof(float) * size_a, hipMemcpyHostToDevice);
  hipMemcpy(GPU_B, B, sizeof(float) * size_b, hipMemcpyHostToDevice);
  
  hipDeviceSynchronize();

  printf("%f %f\n", A[0], GPU_A[0]);
  printf("%f %f\n", B[0], GPU_B[0]);  
  dim3 threadsPerBlock(m,p);
  dim3 blocksPerGrid(1, 1);
  if (size_c > 512){
    threadsPerBlock.x = 512;
    threadsPerBlock.y = 512;
    blocksPerGrid.x = ceil(double(p)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(m)/double(threadsPerBlock.y));
  }

  mat_mul_kernel<<<blocksPerGrid,threadsPerBlock>>>(m,n,p,GPU_A,GPU_B,GPU_C);

  hipDeviceSynchronize();
  // Copy the data back to the host
  hipMemcpy(C, GPU_C, sizeof(float) * size_c, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("%f\n", C[0]);
  hipFree(GPU_A);
  hipFree(GPU_B);
  hipFree(GPU_C);
} 
